
#include <hip/hip_runtime.h>

__device__ float sigmoid(float x) {
    return 1 / (1 + expf(-x));
}

extern "C"
__global__ void produceState2(const float* arguments, const int argsSize, const float* weights, 
                                const int* topology, const int topSize, float* outStates) {
    const int tid = threadIdx.x;
    const int dim = argsSize + topSize;  
    extern __shared__ float s[];
    float* states = s;
    bool* ready = (bool*)&states[dim];
    __shared__ int counter[1]; 

    int r = tid;
    while(r < dim) {
        ready[r] = false;
        r += blockDim.x;
    }        

    
    if (tid == 0) {
        counter[tid] = argsSize;
    }
    if (tid < argsSize) {
        states[tid] = arguments[tid];
        ready[tid] = true;
    }
    __syncthreads();

    while(counter[0] < dim) {
        const int index = counter[0] + tid;
        const int topIndex = index - argsSize;
        if (topIndex < topSize) {
            const int leftBorder = topology[topIndex*3];
            const int rightBorder = topology[topIndex*3 + 1];
            const int weightsStart = topology[topIndex*3 + 2];

            if (rightBorder <= counter[0]) {
                float sum = 0;
                for (int i = leftBorder; i < rightBorder; i++) {
                    sum += states[i] * weights[weightsStart + i - leftBorder];
                }

                states[index] = sigmoid(sum);
                ready[index] = true;
            }
        }
        __syncthreads();

        if (tid == 0) {
            int total = counter[0];
            for (int i = total; i < total + blockDim.x && i < dim; i++) {
                if (ready[i]) {
                    counter[0]++;
                }
            }
        } 
        __syncthreads();
    }

    int n = tid;
    while(n < dim) {
        outStates[n] = states[n];
        n += blockDim.x;
    }
}


extern "C"
__global__ void produceState3(const float* arguments, const int argsSize, const float* weights, 
                                const int* topology, const int topSize, float* outStates) {
    const int tid = threadIdx.x;
    const int dim = argsSize + topSize;  
    extern __shared__ float s[];
    float* states = s;
    int* iters = (int*)&states[dim];      

    if (tid < argsSize) {
        states[tid] = arguments[tid];
        iters[tid] = 1;
    } else {
        iters[tid] = 0;
    }
    __syncthreads();

    while(iters[tid] * blockDim.x + tid < dim) {
        const int index = iters[tid] * blockDim.x + tid;
        const int topIndex = index - argsSize;
        const int leftBorder = topology[topIndex*3];
        const int rightBorder = topology[topIndex*3 + 1];
        const int weightsStart = topology[topIndex*3 + 2];

        bool canStart = true;
        for (int i = leftBorder; i < rightBorder; i++) {
            int threadId = i % blockDim.x;
            int mustCounted = i / blockDim.x + 1;
            if (iters[threadId] < mustCounted) {
                canStart = false;
                break;
            }
        }

        if (canStart) {
            float sum = 0;
            for (int i = leftBorder; i < rightBorder; i++) {
                sum += states[i] * weights[weightsStart + i - leftBorder];
            }
            states[index] = sigmoid(sum);
            iters[tid]++;
        }
        __syncthreads();
    }

    __syncthreads();

    int n = tid;
    while(n < dim) {
        outStates[n] = states[n];
        n += blockDim.x;
    }
}